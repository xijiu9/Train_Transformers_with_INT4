#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <torch/torch.h>
#include <ATen/ATen.h>
#include <cmath>
#include <iostream>
#include <sstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <ctime>
#include "cuda_fp16.hpp"
#include "hip/hip_fp16.h"

#include "hip/hip_runtime.h"
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include <hiprand/hiprand_kernel.h>
#include <tuple>
#include <bits/stdc++.h>
// #include <torch/distributions/gumbel.h>

#include "torch/script.h"
using namespace torch::indexing;

template<typename scalar_t>
__global__ void quantize_cuda_kernel(const scalar_t * __restrict__  MatI, int8_t * first_transform, int8_t * second_transform, 
                                    const int num_bins_half, const int num_bins_clamp, const float scale, long long int size, unsigned long seed){
    long long int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x<size){
        // set random value
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(seed, x, 0, &state);
        const float noise = hiprand_uniform(&state);

        float trans_input = MatI[x] * scale;

        float tmp1 = round(trans_input / num_bins_half);
        int firstTransform = std::clamp((int)(tmp1), -num_bins_clamp, num_bins_clamp);
        first_transform[x] = firstTransform;
        // float quantize = (transform + 8) / scale + zero_point;
        // first_quantize[x] = firstTransform * num_bins_half / scale;

        float tmp2 = round(trans_input - firstTransform * num_bins_half + noise - 0.5);
        int secondTransform = std::clamp((int)(tmp2), -num_bins_clamp, num_bins_clamp);
        second_transform[x] = secondTransform;
        // second_quantize[x] = secondTransform / scale;
    }
}

__global__ void pack_cuda_kernel(int8_t * in, int8_t * out, long long int size){
    long long int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x<size){
        out[x] = (in[(x<<1)+1] << 4) | (in[x<<1] & 15);
    }
}

template<typename scalar_t>
__global__ void multiple_kernel(const scalar_t * __restrict__ in, scalar_t * __restrict__ out, float scale, long long int size){
    long long int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x<size){
        out[x] = in[x] * scale;
    }
}

/// Define a CUTLASS GEMM template and launch a GEMM kernel.
hipError_t CutlassSgemmNN(
  const int M,
  const int N,
  const int K,
  const cutlass::int4b_t *A,
  int lda,
  const cutlass::int4b_t *B,
  int ldb,
  int32_t *C,
  int ldc) {

// The code section below describes datatype for input, output matrices and computation between
// elements in input matrices.
using ElementAccumulator = int32_t;                 // <- data type of accumulator
using ElementComputeEpilogue = ElementAccumulator;  // <- data type of epilogue operations
using ElementInputA = cutlass::int4b_t;                       // <- data type of elements in input matrix A
using ElementInputB = cutlass::int4b_t;                       // <- data type of elements in input matrix B
using ElementOutput = int32_t;                      // <- data type of elements in output matrix D

// The code section below describes matrix layout of input and output matrices. Column Major for
// Matrix A, Row Major for Matrix B and Row Major for Matrix C
using LayoutInputA = cutlass::layout::RowMajor;
using LayoutInputB = cutlass::layout::ColumnMajor;
using LayoutOutput = cutlass::layout::RowMajor;

// This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
using MMAOp = cutlass::arch::OpClassTensorOp;

// This code section describes CUDA SM architecture number
using SmArch = cutlass::arch::Sm80;

// This code section describes the tile size a thread block will compute
using ShapeMMAThreadBlock =
    cutlass::gemm::GemmShape<128, 128, 128>;  // <- threadblock tile M = 128, N = 256, K = 64
// This code section describes tile size a warp will compute
using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 128>;  // <- warp tile M = 64, N = 64, K = 64 
// This code section describes the size of MMA op
using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 64>;  // <- MMA Op tile M = 8, N = 8, K = 16

// This code section describes how threadblocks are scheduled on GPU
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>;  // <- ??

// This code section describes the epilogue part of the kernel
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // <- data type of output matrix
    4,  // <- the number of elements per vectorized
                                                       // memory access. For a byte, it's 16
                                                       // elements. This becomes the vector width of
                                                       // math instructions in the epilogue too
    ElementAccumulator,                                // <- data type of accumulator
    ElementComputeEpilogue>;  // <- data type for alpha/beta in linear combination function

// Number of pipelines you want to use
constexpr int NumStages = 3;

using Gemm = cutlass::gemm::device::Gemm<ElementInputA,
                                         LayoutInputA,
                                         ElementInputB,
                                         LayoutInputB,
                                         ElementOutput,
                                         LayoutOutput,
                                         ElementAccumulator,
                                         MMAOp,
                                         SmArch,
                                         ShapeMMAThreadBlock,
                                         ShapeMMAWarp,
                                         ShapeMMAOp,
                                         EpilogueOp,
                                         SwizzleThreadBlock,
                                         NumStages>;
  // Create a tuple of problem size for matrix multiplication
  cutlass::gemm::GemmCoord problem_size(M, N, K);

  // Initialize alpha and beta for dot product computation
  ElementComputeEpilogue alpha = ElementComputeEpilogue(1);
  ElementComputeEpilogue beta = ElementComputeEpilogue(0);
    
  // Split K dimension into 1 partitions
  int split_k_slices = 1;

  // Create a tuple of gemm kernel arguments. This is later passed as arguments to launch
  // instantiated CUTLASS kernel
  typename Gemm::Arguments arguments{problem_size,  // <- problem size of matrix multiplication
                                     {A, lda},  // <- reference to matrix A on device
                                     {B, ldb},  // <- reference to matrix B on device
                                     {C, ldc},  // <- reference to matrix C on device
                                     {C, ldc},  // <- reference to matrix D on device
                                     {alpha, beta},          // <- tuple of alpha and beta
                                     split_k_slices};        // <- k-dimension split factor
  
    Gemm gemm_op;
    cutlass::Status status = gemm_op(arguments);

  if (status != cutlass::Status::kSuccess) {
    return hipErrorUnknown;
  }

  // Return success, if no errors were encountered.
  return hipSuccess;
}

#define N_THREADS 256

template<typename scalar_t>
__global__ void dequantize_cuda_kernel(const int32_t * gemm1, const int32_t * gemm2, const scalar_t * __restrict__ gemm3, 
                                        const scalar_t * __restrict__ gemm4, scalar_t * __restrict__ output, 
                                        const float scale_gemm1, const float scale_gemm2, long long int size){  
    long long int x = threadIdx.x + blockIdx.x * blockDim.x;
    // int row = x / ny, col = x - row * ny;

    if (x<size){
       output[x] = gemm1[x] * scale_gemm1 + gemm2[x] * scale_gemm2 + gemm3[x] + gemm4[x];
        // output[x] = 0;
    }
}

template<typename scalar_t>
__global__ void dequantize2_cuda_kernel(const int32_t * gemm1, const int32_t * gemm2, scalar_t * __restrict__ output, 
                                        const float scale_gemm1, const float scale_gemm2, long long int size){  
    long long int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x<size){
       output[x] = gemm1[x] * scale_gemm1 + gemm2[x] * scale_gemm2;
    }
}

template<typename scalar_t>
__global__ void LSQ_cuda_kernel(const scalar_t * lsq_weight, const scalar_t * __restrict__ grad_output, scalar_t * __restrict__ grad_alpha_out, 
                                scalar_t * __restrict__ grad_input, const float grad_scale, const long long int size){  
    long long int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x<size){
       scalar_t q_w = lsq_weight[x];
       scalar_t indicate_small = (q_w < -8);
       scalar_t indicate_big = (q_w > 7);
       scalar_t indicate_middle = 1.0 - indicate_small - indicate_big;
       scalar_t grad_out = grad_output[x];
       grad_alpha_out[x] = (indicate_small * -8 + indicate_big * 7 + indicate_middle * (-q_w + round(q_w))) * grad_out * grad_scale;
       grad_input[x] = indicate_middle * grad_out;
    }
}


__device__ __inline__ c10::Half __shfl_down_sync(const unsigned mask, const c10::Half var,
                                                 const unsigned int delta, const int width) {
  __half var_ = var;
  return __shfl_down_sync(mask, var_, delta, width);
}

__device__ __inline__ c10::Half __shfl_sync(const unsigned mask, const c10::Half var,
                                            const int delta, const int width) {
  __half var_ = var;
  return __shfl_sync(mask, var_, delta, width);
}

template <typename scalar_t>
__global__ void minimax_cuda_kernel(const scalar_t* __restrict__ data,
                                    scalar_t* __restrict__ min,
                                    scalar_t* __restrict__ max,
                                    int64_t N,
                                    int64_t D) {
  scalar_t max_val, min_val;
  max_val = -1e30;
  min_val = 1e30;

  for (int64_t k1_outer = 0; k1_outer < D / 32; ++k1_outer) {
    max_val = std::max(max_val, data[blockIdx.x * D + k1_outer * 32 + threadIdx.x]);
    min_val = std::min(min_val, data[blockIdx.x * D + k1_outer * 32 + threadIdx.x]);
  }

  unsigned int mask;
  scalar_t max_val_t, min_val_t;
  mask = __activemask();

  max_val_t = __shfl_down_sync(mask, max_val, 16, 32);
  max_val = std::max(max_val, max_val_t);
  max_val_t = __shfl_down_sync(mask, max_val, 8, 32);
  max_val = std::max(max_val, max_val_t);
  max_val_t = __shfl_down_sync(mask, max_val, 4, 32);
  max_val = std::max(max_val, max_val_t);
  max_val_t = __shfl_down_sync(mask, max_val, 2, 32);
  max_val = std::max(max_val, max_val_t);
  max_val_t = __shfl_down_sync(mask, max_val, 1, 32);
  max_val = std::max(max_val, max_val_t);
  max_val = __shfl_sync(mask, max_val, 0, 32);
  max[blockIdx.x] = max_val;

  min_val_t = __shfl_down_sync(mask, min_val, 16, 32);
  min_val = std::min(min_val, min_val_t);
  min_val_t = __shfl_down_sync(mask, min_val, 8, 32);
  min_val = std::min(min_val, min_val_t);
  min_val_t = __shfl_down_sync(mask, min_val, 4, 32);
  min_val = std::min(min_val, min_val_t);
  min_val_t = __shfl_down_sync(mask, min_val, 2, 32);
  min_val = std::min(min_val, min_val_t);
  min_val_t = __shfl_down_sync(mask, min_val, 1, 32);
  min_val = std::min(min_val, min_val_t);
  min_val = __shfl_sync(mask, min_val, 0, 32);
  min[blockIdx.x] = min_val;
}

//TODO: N means rows, D means cols
template<typename scalar_t>
__global__ void linalg_norm_cuda_kernel(const scalar_t * __restrict__ in, float * linalg, int N, int D, int stride_D){
  float sum_val = 0;

  for (int64_t k1_outer = 0; k1_outer < stride_D; ++k1_outer) {
    float temp = in[blockIdx.x * D + (k1_outer << 5) + threadIdx.x];
    sum_val += temp * temp;
  }

  unsigned int mask;
  float sum_val_t;
  mask = __activemask();

  sum_val_t = __shfl_down_sync(mask, sum_val, 16, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 8, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 4, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 2, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 1, 32);
  sum_val += sum_val_t;
  linalg[blockIdx.x] = sqrt(sum_val);
}

__global__ void linalg_normInt_cuda_kernel(const int8_t * in, float * linalg, int N, int D, int stride_D, float scale){
  float sum_val = 0;

  for (int64_t k1_outer = 0; k1_outer < stride_D; ++k1_outer) {
    int64_t temp = in[blockIdx.x * D + (k1_outer << 5) + threadIdx.x];
    sum_val += temp * temp;
  }

  unsigned int mask;
  float sum_val_t;
  mask = __activemask();

  sum_val_t = __shfl_down_sync(mask, sum_val, 16, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 8, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 4, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 2, 32);
  sum_val += sum_val_t;
  sum_val_t = __shfl_down_sync(mask, sum_val, 1, 32);
  sum_val += sum_val_t;
  linalg[blockIdx.x] = sqrt(sum_val) * scale;
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, std::vector<double>, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, float> quantize_cuda(torch::Tensor x, int num_bits, torch::Tensor qy, float scaley, torch::Tensor lsq_weight){
    std::vector<double> time_vector;
    long long int nz = x.size(0);
    long long int nx = x.size(1);
    long long int ny = qy.size(1);

    hipDeviceSynchronize();
    clock_t time_quantize_start = clock();

    auto option_transform = torch::TensorOptions().dtype(torch::kInt8).device(x.device());
    auto option_quantize = torch::TensorOptions().dtype(x.dtype()).device(x.device());
    auto option_float = torch::TensorOptions().dtype(torch::kFloat32).device(x.device());
    torch::Tensor first_transform = torch::empty({nz, nx}, option_transform);
    // torch::Tensor first_quantize = torch::empty({nz, nx}, option_quantize);
    torch::Tensor second_transform = torch::empty({nz, nx}, option_transform);
    // torch::Tensor second_quantize = torch::empty({nz, nx}, option_quantize);

    dim3 block(N_THREADS);
    dim3 grid1((nx*nz-1)/block.x+1);
    long long int size_quantize = nz * nx ;
    // process of first quantize
    torch::Tensor min_x = torch::empty({nz, }, option_quantize);
    torch::Tensor max_x = torch::empty({nz, }, option_quantize);
    int minimax_blocks = nz;
    int minimax_threads = 32;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "minimax_cuda", ([&] {
    minimax_cuda_kernel<scalar_t><<<minimax_blocks, minimax_threads>>>(
      x.data_ptr<scalar_t>(), min_x.data_ptr<scalar_t>(), max_x.data_ptr<scalar_t>(),
      nz, nx);
    }));
    // float mn = std::min(x.min().item<float>() - 1e-8, 0.);
    // float mx = std::max(x.max().item<float>() + 1e-8, 0.);
    float mn = std::min(min_x.min().item<float>() - 1e-8, 0.);
    float mx = std::max(max_x.max().item<float>() + 1e-8, 0.);

    int num_bins_half = pow(2, num_bits) - 2;
    int num_bins = num_bins_half * num_bins_half;
    int num_bins_clamp = num_bins_half / 2 - 1;

    float scale1 = num_bins / (2 * max(fabs(mn), fabs(mx)));

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(x.scalar_type(), "quantize_cuda", ([&] {
    quantize_cuda_kernel<scalar_t><<<grid1, block>>>(
        x.data_ptr<scalar_t>(),
        first_transform.data_ptr<int8_t>(),
        // first_quantize.data_ptr<scalar_t>(),
        second_transform.data_ptr<int8_t>(),
        // second_quantize.data_ptr<scalar_t>(),
        num_bins_half, num_bins_clamp,
        scale1, size_quantize,rand());
    }));

    hipDeviceSynchronize();
    clock_t time_quantize_end = clock();

    // leverage score
    // TODO: use dim=0 because torch.linalg only supports dim=1
    int threads = 32;
    int blocks = nz;

    auto x1_len = torch::empty({nz,}, option_float);
    auto x2_len = torch::empty({nz,}, option_float);
    auto y_len = torch::empty({nz,}, option_float);

    int stride_x = nx / 32;
    float scale_x1 = num_bins_half / scale1;
    float scale_x2 = 1. / scale1;
    // AT_DISPATCH_FLOATING_TYPES_AND_HALF(first_quantize.scalar_type(), "linalg_cuda", ([&] {
    // linalg_norm_cuda_kernel<scalar_t><<<blocks, threads>>>(
    //     first_quantize.data_ptr<scalar_t>(), 
    //     x1_len.data_ptr<float>(),
    //     nz,nx,stride_x);
    // }));
    // AT_DISPATCH_FLOATING_TYPES_AND_HALF(second_quantize.scalar_type(), "linalg_cuda", ([&] {
    // linalg_norm_cuda_kernel<scalar_t><<<blocks, threads>>>(
    //     second_quantize.data_ptr<scalar_t>(), 
    //     x2_len.data_ptr<float>(),
    //     nz,nx,stride_x);
    // }));

    linalg_normInt_cuda_kernel<<<blocks, threads>>>(
        first_transform.data_ptr<int8_t>(), 
        x1_len.data_ptr<float>(),
        nz,nx,stride_x, scale_x1);

    linalg_normInt_cuda_kernel<<<blocks, threads>>>(
        second_transform.data_ptr<int8_t>(), 
        x2_len.data_ptr<float>(),
        nz,nx,stride_x, scale_x2);

    int stride_y = ny / 32;
    // AT_DISPATCH_FLOATING_TYPES_AND_HALF(y.scalar_type(), "linalg_cuda", ([&] {
    // linalg_norm_cuda_kernel<scalar_t><<<blocks, threads>>>(
    //     y.data_ptr<scalar_t>(), 
    //     y_len.data_ptr<float>(),
    //     nz,ny,stride_y);
    // }));
    linalg_normInt_cuda_kernel<<<blocks, threads>>>(
        qy.data_ptr<int8_t>(), 
        y_len.data_ptr<float>(),
        nz,ny,stride_y, scaley);

    // TODO: whether need to change dtype from half into float? It depends 
    auto vec_norm = torch::cat({torch::mul(x1_len, y_len), torch::mul(x2_len, y_len)});
    // auto vec_norm = torch::cat({torch::mul(x1_len, y_len), torch::mul(x2_len, y_len)});
    int len_norm = vec_norm.numel();

    hipDeviceSynchronize();
    clock_t time_leverage_end = clock();

    int cnt = 0;
    int flag = 0;
    // auto norm_weight_loop = vec_norm * len_norm / (2 * vec_norm.sum());
    auto norm_weight_loop = torch::empty_like(vec_norm);
    float scale_norm = len_norm / (2 * vec_norm.sum().item<float>());
    dim3 grid_norm(len_norm/block.x+1);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(vec_norm.scalar_type(), "multiple_cuda", ([&] {
    multiple_kernel<scalar_t><<<grid_norm, block>>>(
        vec_norm.data_ptr<scalar_t>(),
        norm_weight_loop.data_ptr<scalar_t>(),
        scale_norm,len_norm);
    }));
    auto sample_index = torch::empty_like(norm_weight_loop);
    int posNum = (norm_weight_loop > 0).sum().item<int>();
    // if (posNum < len_norm / 2){
    if (true) {
    // if (false) {
        cnt = posNum;
        norm_weight_loop.index_put_({norm_weight_loop > 0}, 1);
        sample_index = norm_weight_loop;
        flag = 2;
    }else{
        bool whileloop = (norm_weight_loop.max() > 1).item<bool>();
        while (1){
            if (!(whileloop && cnt < len_norm / 2)) {
                flag = 1;
                break;
            }
            auto small_index = (norm_weight_loop < 1);
            auto small_value = norm_weight_loop.index({small_index});
            long long int small_len = small_value.numel();
            cnt = len_norm - small_len;
            norm_weight_loop = torch::clamp(norm_weight_loop, 0, 1);
            bool breakloop = (small_value.max() == 0).item<bool>();
            if (breakloop) {
                flag = 2;
                break;
            }
            // small_value = small_value * (len_norm / 2 - cnt) / small_value.sum();
            float scale_small = (len_norm / 2 - cnt) / small_value.sum().item<float>();
            dim3 grid_small(small_len/block.x+1);
            AT_DISPATCH_FLOATING_TYPES_AND_HALF(small_value.scalar_type(), "multiple_cuda", ([&] {
            multiple_kernel<scalar_t><<<grid_small, block>>>(
                small_value.data_ptr<scalar_t>(),
                small_value.data_ptr<scalar_t>(),
                scale_small,small_len);
            }));
            // norm_weight_loop[small_index] = small_value;
            norm_weight_loop.index_put_({small_index}, small_value);
            whileloop = (norm_weight_loop.max() > 1).item<bool>();
        } 
        sample_index = torch::bernoulli(norm_weight_loop);
    }
    // auto sample_index = torch::bernoulli(norm_weight_loop);
    auto small_indices = torch::nonzero(sample_index.index({Slice({None, len_norm/2})}) == 1).squeeze(1);
    auto large_indices = torch::nonzero(sample_index.index({Slice(len_norm/2)}) == 1).squeeze(1);

    auto option_output = torch::TensorOptions().dtype(x.dtype()).device(x.device());
    auto grad_output = torch::empty({nx,ny}, option_output);
    dim3 grid2((nx*ny-1)/block.x+1);
    long long int size = nx*ny;

    clock_t time_sampleflag_end;
    clock_t time_sample_end;
    clock_t time_pack_end;
    clock_t time_gemm_end;
    clock_t time_dequantize_end;
    hipDeviceSynchronize();
    clock_t time_method_start = clock();
    int small_num_, large_num_;

    if (flag == 1){
        auto norm_small_indices = (norm_weight_loop.index({small_indices}) == 1);
        auto small_num = norm_small_indices.sum();
        // TODO: test if .int() can work in libtorch
        small_num = ((small_num / 32).floor() * 32).to(torch::kInt32);
        small_num_ = small_num.item<int>();
        auto small_int_indices = small_indices.index({norm_small_indices}).index({Slice({None, small_num_})});
        auto small_left_indices = small_indices.index({~torch::isin(small_indices, small_int_indices)});
        auto norm_large_indices = (norm_weight_loop.index({large_indices + len_norm / 2}) == 1);
        auto large_num = norm_large_indices.sum();
        // TODO: test if .int() can work in libtorch
        large_num = ((large_num / 32).floor() * 32).to(torch::kInt32);
        large_num_ = large_num.item<int>();
        auto large_int_indices = large_indices.index({norm_large_indices}).index({Slice({None, large_num_})});
        auto large_left_indices = large_indices.index({~torch::isin(large_indices, large_int_indices)});
        // auto _index = torch::nonzero((sample_index == 1)).squeeze();

        norm_weight_loop.index_put_({norm_weight_loop == 0}, 1e-10);

        
        // auto small_num_ = (_index < len_norm / 2).sum();
        // auto large_num_ = _index.numel() - small_num_;
        // auto small_indices = _index.index({Slice({None, small_num_.item<int>()})});
        // auto large_indices = _index.index({Slice(small_num_.item<int>())}) - int(len_norm / 2);
        // auto norm_weight_small = norm_weight_loop.index({small_indices});
        // auto norm_weight_large = norm_weight_loop.index({large_indices + len_norm / 2});
        // auto output = torch::cat({first_quantize, second_quantize});
        // output = output / norm_weight_loop.unsqueeze(1);
        hipDeviceSynchronize();
        time_sampleflag_end = clock();


        //TODO: suppose an easy situation so that it can be faster
        auto sample_x1 = first_transform.index({small_int_indices}).t().contiguous();
        auto sample_x2 = second_transform.index({large_int_indices}).t().contiguous();
        auto sample_y1 = qy.index({small_int_indices}).t().contiguous();
        auto sample_y2 = qy.index({large_int_indices}).t().contiguous();
        auto sample_x3 = (first_transform.index({small_left_indices}).t() * num_bins_half / (scale1 * norm_weight_loop.index({small_left_indices}))).to(x.dtype());
        auto sample_x4 = (second_transform.index({large_left_indices}).t() / (scale1 * norm_weight_loop.index({large_left_indices + len_norm / 2}))).to(x.dtype());
        //todo:currently multiply a scaley to convert it into fp16
        auto sample_y3 = (qy.index({small_left_indices}) * scaley).to(x.dtype());
        auto sample_y4 = (qy.index({large_left_indices}) * scaley).to(x.dtype());

        hipDeviceSynchronize();
        time_sample_end = clock();

        // pack process
        // auto option_transform = torch::TensorOptions().dtype(torch::kInt8).device(x.device());
        auto sample_x1_int4 = torch::empty({nx, small_num_>>1}, option_transform);
        auto sample_x2_int4 = torch::empty({nx, large_num_>>1}, option_transform);
        auto sample_y1_int4 = torch::empty({ny, small_num_>>1}, option_transform);
        auto sample_y2_int4 = torch::empty({ny, large_num_>>1}, option_transform);
        long long int grid_size_x1 = nx*small_num_/2;
        long long int grid_size_x2 = nx*large_num_/2;
        long long int grid_size_y1 = ny*small_num_/2;
        long long int grid_size_y2 = ny*large_num_/2;
        dim3 grid_pack_x1((grid_size_x1-1)/block.x+1);
        dim3 grid_pack_x2((grid_size_x2-1)/block.x+1);
        dim3 grid_pack_y1((grid_size_y1-1)/block.x+1);
        dim3 grid_pack_y2((grid_size_y2-1)/block.x+1);
        if (small_num_ > 0) {
            pack_cuda_kernel<<<grid_pack_x1,block>>>(sample_x1.data_ptr<int8_t>(), sample_x1_int4.data_ptr<int8_t>(), grid_size_x1);
            pack_cuda_kernel<<<grid_pack_y1,block>>>(sample_y1.data_ptr<int8_t>(), sample_y1_int4.data_ptr<int8_t>(), grid_size_y1);
        }
        if (large_num_ > 0) {
            pack_cuda_kernel<<<grid_pack_x2,block>>>(sample_x2.data_ptr<int8_t>(), sample_x2_int4.data_ptr<int8_t>(), grid_size_x2);
            pack_cuda_kernel<<<grid_pack_y2,block>>>(sample_y2.data_ptr<int8_t>(), sample_y2_int4.data_ptr<int8_t>(), grid_size_y2);
        }

        hipDeviceSynchronize();
        time_pack_end = clock();

        // gemm process
        hipError_t result;
        int lda_first = small_num_;
        int ldb_first = small_num_;
        int ldc = ny;
        // Chunked matrix multiplication
        auto gemm1 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
        if (small_num_ > 0) {
            result = CutlassSgemmNN(nx, ny, small_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x1_int4.data_ptr<int8_t>()), lda_first, reinterpret_cast<cutlass::int4b_t *>(sample_y1_int4.data_ptr<int8_t>()), ldb_first, gemm1.data_ptr<int32_t>(), ldc);
        } else {
            gemm1 = torch::zeros({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
        }
        // result = CutlassSgemmNN(nx, ny, small_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x1_int4.data_ptr<int8_t>()), lda_first, reinterpret_cast<cutlass::int4b_t *>(sample_y1_int4.data_ptr<int8_t>()), ldb_first, gemm1.data_ptr<int32_t>(), ldc);

        int lda_second = large_num_;
        int ldb_second = large_num_;
        auto gemm2 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
        if (large_num_ > 0) {
            result = CutlassSgemmNN(nx, ny, large_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x2_int4.data_ptr<int8_t>()), lda_second, reinterpret_cast<cutlass::int4b_t *>(sample_y2_int4.data_ptr<int8_t>()), ldb_second, gemm2.data_ptr<int32_t>(), ldc);
        } else {
            gemm2 = torch::zeros({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
        }
        // result = CutlassSgemmNN(nx, ny, large_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x2_int4.data_ptr<int8_t>()), lda_second, reinterpret_cast<cutlass::int4b_t *>(sample_y2_int4.data_ptr<int8_t>()), ldb_second, gemm2.data_ptr<int32_t>(), ldc);

        // hipDeviceSynchronize();
        // clock_t time_int4gemm_end = clock();

        auto gemm3 = torch::matmul(sample_x3, sample_y3);
        auto gemm4 = torch::matmul(sample_x4, sample_y4);
        // auto gemm3 = torch::empty({nx, ny}, at::device(at::kCUDA).dtype(torch::kFloat16));
        // auto gemm4 = torch::empty({nx, ny}, at::device(at::kCUDA).dtype(torch::kFloat16));


        // auto gemm3 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kFloat16));

        // auto gemm2 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kFloat16));

        // hipDeviceSynchronize();
        // clock_t time_fp16gemm_end = clock();
        hipDeviceSynchronize();
        time_gemm_end = clock();

        // dequantize process
        // First dequantize higher 4 bits
        // auto sum_y1_column = torch::sum(qy.index({small_int_indices}), 0);
        // auto sum_y2_column = torch::sum(qy.index({large_int_indices}), 0);

        // float const_x1 = (8.0 / scale1 + zero_point1) * scaley;
        // float const_x2 = (8.0 / scale2 + zero_point2) * scaley;
        // float const_x = const_x1 + const_x2;
        float scale_gemm1 = scaley * num_bins_half / (scale1);
        float scale_gemm2 = scaley / (scale1);
        AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "dequantize_cuda", ([&] {
        dequantize_cuda_kernel<scalar_t><<<grid2, block>>>(
            gemm1.data_ptr<int32_t>(), 
            gemm2.data_ptr<int32_t>(),
            gemm3.data_ptr<scalar_t>(), 
            gemm4.data_ptr<scalar_t>(),
            grad_output.data_ptr<scalar_t>(),
            // sum_y1_column.data_ptr<int64_t>(),
            // sum_y2_column.data_ptr<int64_t>(),
            // const_x1, const_x2, 
            scale_gemm1, scale_gemm2,
            size);
        }));

        hipDeviceSynchronize();
        time_dequantize_end = clock();

    } else if (flag == 2){
        // if (cnt > 800){
        if (false) {
            small_num_ = floor(small_indices.numel() / 32.0) * 32;
            large_num_ = floor(large_indices.numel() / 32.0) * 32;

            auto small_int_indices = small_indices.index({Slice({None, small_num_})});
            auto large_int_indices = large_indices.index({Slice({None, large_num_})});
            hipDeviceSynchronize();
            time_sampleflag_end = clock();

            auto sample_x1 = first_transform.index({small_int_indices}).t().contiguous();
            auto sample_x2 = second_transform.index({large_int_indices}).t().contiguous();
            auto sample_y1 = qy.index({small_int_indices}).t().contiguous();
            auto sample_y2 = qy.index({large_int_indices}).t().contiguous();

            hipDeviceSynchronize();
            time_sample_end = clock();

            auto sample_x1_int4 = torch::empty({nx, small_num_>>1}, option_transform);
            auto sample_x2_int4 = torch::empty({nx, large_num_>>1}, option_transform);
            auto sample_y1_int4 = torch::empty({ny, small_num_>>1}, option_transform);
            auto sample_y2_int4 = torch::empty({ny, large_num_>>1}, option_transform);
            long long int grid_size_x1 = nx*small_num_/2;
            long long int grid_size_x2 = nx*large_num_/2;
            long long int grid_size_y1 = ny*small_num_/2;
            long long int grid_size_y2 = ny*large_num_/2;
            dim3 grid_pack_x1((grid_size_x1-1)/block.x+1);
            dim3 grid_pack_x2((grid_size_x2-1)/block.x+1);
            dim3 grid_pack_y1((grid_size_y1-1)/block.x+1);
            dim3 grid_pack_y2((grid_size_y2-1)/block.x+1);
            if (small_num_ > 0) {
                pack_cuda_kernel<<<grid_pack_x1,block>>>(sample_x1.data_ptr<int8_t>(), sample_x1_int4.data_ptr<int8_t>(), grid_size_x1);
                pack_cuda_kernel<<<grid_pack_y1,block>>>(sample_y1.data_ptr<int8_t>(), sample_y1_int4.data_ptr<int8_t>(), grid_size_y1);
            }
            if (large_num_ > 0) {
                pack_cuda_kernel<<<grid_pack_x2,block>>>(sample_x2.data_ptr<int8_t>(), sample_x2_int4.data_ptr<int8_t>(), grid_size_x2);
                pack_cuda_kernel<<<grid_pack_y2,block>>>(sample_y2.data_ptr<int8_t>(), sample_y2_int4.data_ptr<int8_t>(), grid_size_y2);
            }
            hipDeviceSynchronize();
            time_pack_end = clock();

            hipError_t result;
            int lda_first = small_num_;
            int ldb_first = small_num_;
            int ldc = ny;
            // Chunked matrix multiplication
            auto gemm1 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
            if (small_num_ > 0) {
                result = CutlassSgemmNN(nx, ny, small_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x1_int4.data_ptr<int8_t>()), lda_first, reinterpret_cast<cutlass::int4b_t *>(sample_y1_int4.data_ptr<int8_t>()), ldb_first, gemm1.data_ptr<int32_t>(), ldc);
            } else {
                gemm1 = torch::zeros({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
            }

            int lda_second = large_num_;
            int ldb_second = large_num_;
            auto gemm2 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
            if (large_num_ > 0) {
                result = CutlassSgemmNN(nx, ny, large_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x2_int4.data_ptr<int8_t>()), lda_second, reinterpret_cast<cutlass::int4b_t *>(sample_y2_int4.data_ptr<int8_t>()), ldb_second, gemm2.data_ptr<int32_t>(), ldc);
            } else {
                gemm2 = torch::zeros({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
            }
            hipDeviceSynchronize();
            time_gemm_end = clock();

            // First dequantize higher 4 bits
            float scale_gemm1 = scaley * num_bins_half / (scale1);
            float scale_gemm2 = scaley / (scale1);
            AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "dequantize2_cuda", ([&] {
            dequantize2_cuda_kernel<scalar_t><<<grid2, block>>>(
                gemm1.data_ptr<int32_t>(), 
                gemm2.data_ptr<int32_t>(),
                grad_output.data_ptr<scalar_t>(),
                scale_gemm1, scale_gemm2,
                size);
            }));
            hipDeviceSynchronize();
            time_dequantize_end = clock();
        } else{
            small_num_ = ceil(small_indices.numel() / 32.0) * 32;
            large_num_ = ceil(large_indices.numel() / 32.0) * 32;

            // int small_pad = small_num_ - small_indices.numel();
            // if (small_pad > 0) {
            //     auto small_pad_indices = (torch::nonzero(sample_index.index({Slice({None, len_norm/2})}) != 1).squeeze(1)).index({Slice({None, small_pad})});
            //     if (small_pad_indices.numel() < small_pad) exit(0);
            //     small_indices = torch::cat({small_indices, small_pad_indices}, 0);
            // }
            // int large_pad = large_num_ - large_indices.numel();
            // if (large_pad > 0) {
            //     auto large_pad_indices = (torch::nonzero(sample_index.index({Slice(len_norm/2)}) != 1).squeeze(1)).index({Slice({None, large_pad})}); 
            //     if (large_pad_indices.numel() < large_pad) exit(0);
            //     large_indices = torch::cat({large_indices, large_pad_indices}, 0);
            // }
            

            // // std::vector<int64_t> padding_small = {0,0,0,small_num_ - small_indices.numel()};
            // // std::vector<int64_t> padding_large = {0,0,0,large_num_ - large_indices.numel()};

            // // torch::nn::ZeroPad2d pad_small(padding_small);
            // // torch::nn::ZeroPad2d pad_large(padding_large);
            // hipDeviceSynchronize();
            // time_sampleflag_end = clock();

            // auto sample_x1 = first_transform.index({small_indices}).t().contiguous();
            // auto sample_x2 = second_transform.index({large_indices}).t().contiguous();
            // auto sample_y1 = qy.index({small_indices}).t().contiguous();
            // auto sample_y2 = qy.index({large_indices}).t().contiguous();

            std::vector<int64_t> padding_small = {0,0,0,small_num_ - small_indices.numel()};
            std::vector<int64_t> padding_large = {0,0,0,large_num_ - large_indices.numel()};

            torch::nn::ZeroPad2d pad_small(padding_small);
            torch::nn::ZeroPad2d pad_large(padding_large);
            hipDeviceSynchronize();
            time_sampleflag_end = clock();

            auto sample_x1 = pad_small(first_transform.index({small_indices})).t().contiguous();
            auto sample_x2 = pad_large(second_transform.index({large_indices})).t().contiguous();
            auto sample_y1 = pad_small(qy.index({small_indices})).t().contiguous();
            auto sample_y2 = pad_large(qy.index({large_indices})).t().contiguous();

            hipDeviceSynchronize();
            time_sample_end = clock();

            auto sample_x1_int4 = torch::empty({nx, small_num_>>1}, option_transform);
            auto sample_x2_int4 = torch::empty({nx, large_num_>>1}, option_transform);
            auto sample_y1_int4 = torch::empty({ny, small_num_>>1}, option_transform);
            auto sample_y2_int4 = torch::empty({ny, large_num_>>1}, option_transform);
            long long int grid_size_x1 = nx*small_num_/2;
            long long int grid_size_x2 = nx*large_num_/2;
            long long int grid_size_y1 = ny*small_num_/2;
            long long int grid_size_y2 = ny*large_num_/2;
            dim3 grid_pack_x1((grid_size_x1-1)/block.x+1);
            dim3 grid_pack_x2((grid_size_x2-1)/block.x+1);
            dim3 grid_pack_y1((grid_size_y1-1)/block.x+1);
            dim3 grid_pack_y2((grid_size_y2-1)/block.x+1);
            if (small_num_ > 0) {
                pack_cuda_kernel<<<grid_pack_x1,block>>>(sample_x1.data_ptr<int8_t>(), sample_x1_int4.data_ptr<int8_t>(), grid_size_x1);
                pack_cuda_kernel<<<grid_pack_y1,block>>>(sample_y1.data_ptr<int8_t>(), sample_y1_int4.data_ptr<int8_t>(), grid_size_y1);
            }
            if (large_num_ > 0) {
                pack_cuda_kernel<<<grid_pack_x2,block>>>(sample_x2.data_ptr<int8_t>(), sample_x2_int4.data_ptr<int8_t>(), grid_size_x2);
                pack_cuda_kernel<<<grid_pack_y2,block>>>(sample_y2.data_ptr<int8_t>(), sample_y2_int4.data_ptr<int8_t>(), grid_size_y2);
            }

            hipDeviceSynchronize();
            time_pack_end = clock();

            hipError_t result;
            int lda_first = small_num_;
            int ldb_first = small_num_;
            int ldc = ny;
            // Chunked matrix multiplication
            auto gemm1 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
            if (small_num_ > 0) {
                result = CutlassSgemmNN(nx, ny, small_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x1_int4.data_ptr<int8_t>()), lda_first, reinterpret_cast<cutlass::int4b_t *>(sample_y1_int4.data_ptr<int8_t>()), ldb_first, gemm1.data_ptr<int32_t>(), ldc);
            } else {
                gemm1 = torch::zeros({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
            }

            int lda_second = large_num_;
            int ldb_second = large_num_;
            auto gemm2 = torch::empty({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
            if (large_num_ > 0) {
                result = CutlassSgemmNN(nx, ny, large_num_, reinterpret_cast<cutlass::int4b_t *>(sample_x2_int4.data_ptr<int8_t>()), lda_second, reinterpret_cast<cutlass::int4b_t *>(sample_y2_int4.data_ptr<int8_t>()), ldb_second, gemm2.data_ptr<int32_t>(), ldc);
            } else {
                gemm2 = torch::zeros({nx,ny}, at::device(at::kCUDA).dtype(torch::kInt32));
            }
            hipDeviceSynchronize();
            time_gemm_end = clock();

            // First dequantize higher 4 bits
            float scale_gemm1 = scaley * num_bins_half / (scale1);
            float scale_gemm2 = scaley / (scale1);
            AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "dequantize2_cuda", ([&] {
            dequantize2_cuda_kernel<scalar_t><<<grid2, block>>>(
                gemm1.data_ptr<int32_t>(), 
                gemm2.data_ptr<int32_t>(),
                grad_output.data_ptr<scalar_t>(),
                scale_gemm1, scale_gemm2,
                size);
            }));
            hipDeviceSynchronize();
            time_dequantize_end = clock();
        }
        
    }

    // AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "dequantize_cuda", ([&] {
    // dequantize_cuda_kernel_fp16<scalar_t><<<grid2, block>>>(
    //     gemm1.data_ptr<scalar_t>(), 
    //     gemm2.data_ptr<scalar_t>(),
    //     grad_output.data_ptr<scalar_t>(),
    //     size);
    // }));

    float grad_scale = 1.0 / sqrt(lsq_weight.numel() * 7);
    auto grad_alpha_out = torch::empty({nx,ny}, option_output);
    auto grad_input = torch::empty({nx,ny}, option_output);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output.scalar_type(), "LSQ_cuda", ([&] {
    LSQ_cuda_kernel<scalar_t><<<grid2, block>>>(
        lsq_weight.data_ptr<scalar_t>(), 
        grad_output.data_ptr<scalar_t>(),
        grad_alpha_out.data_ptr<scalar_t>(),
        grad_input.data_ptr<scalar_t>(),
        grad_scale, size);
    }));

    // auto q_w = hadamard_weight / scale_weight;
    // auto indicate_small = (q_w < -8).to(torch::kFloat16);
    // auto indicate_big = (q_w > 7).to(torch::kFloat16);
    // auto indicate_middle = 1.0 - indicate_small - indicate_big;
    // auto grad_alpha = ((indicate_small * -8 + indicate_big * 7 + indicate_middle * (
    //                 -q_w + q_w.round())) * grad_output * grad_scale).sum().unsqueeze(0);
    // auto grad_input = indicate_middle * grad_output; 
    auto grad_alpha = grad_alpha_out.sum().unsqueeze(0);
    //TODO:to test use this way, later change into Gumble

    hipDeviceSynchronize();
    clock_t time_LSQ_end = clock();

    // double quantize1_time = (double)(time_quantize1_end - time_quantize1_start) / CLOCKS_PER_SEC;
    double quantize_time = (double)(time_quantize_end - time_quantize_start) / CLOCKS_PER_SEC;
    double leverage_time = (double)(time_leverage_end - time_quantize_end) / CLOCKS_PER_SEC;
    double sample_time = (double)(time_sample_end - time_leverage_end) / CLOCKS_PER_SEC;
    // double sample1_time = (double)(time_method_start - time_leverage_end) / CLOCKS_PER_SEC;
    // double sample2_time = (double)(time_sampleflag_end - time_method_start) / CLOCKS_PER_SEC;
    // double sample3_time = (double)(time_sample_end - time_sampleflag_end) / CLOCKS_PER_SEC;
    double pack_time = (double)(time_pack_end - time_sample_end) / CLOCKS_PER_SEC;
    double gemm_time = (double)(time_gemm_end - time_pack_end) / CLOCKS_PER_SEC;
    double dequantize_time = (double)(time_dequantize_end - time_gemm_end) / CLOCKS_PER_SEC;
    double method1_time= 0, method2_time = 0, method3_time = 0;
    if (flag == 1){
        method1_time = (double)(time_dequantize_end - time_method_start) / CLOCKS_PER_SEC;
    }else if (flag == 2){
        if (cnt > 800){
            method2_time = (double)(time_dequantize_end - time_method_start) / CLOCKS_PER_SEC;
        } else{
            method3_time = (double)(time_dequantize_end - time_method_start) / CLOCKS_PER_SEC;
        }
    }
    // double pack_time = (double)(time_pack_end - time_sample_end) / CLOCKS_PER_SEC;
    // double int4gemm_time = (double)(time_int4gemm_end - time_pack_end) / CLOCKS_PER_SEC;
    // double fp16gemm_time = (double)(time_fp16gemm_end - time_int4gemm_end) / CLOCKS_PER_SEC;
    // double dequantize_time = (double)(time_dequantize_end - time_fp16gemm_end) / CLOCKS_PER_SEC;
    double LSQ_time = (double)(time_LSQ_end - time_dequantize_end) / CLOCKS_PER_SEC;
    // // time_leverage_end

    // time_vector.push_back(quantize1_time);
    time_vector.push_back(quantize_time);
    time_vector.push_back(leverage_time);
    time_vector.push_back(sample_time);
    // time_vector.push_back(sample1_time);
    // time_vector.push_back(sample2_time);
    // time_vector.push_back(sample3_time);
    time_vector.push_back(pack_time);
    time_vector.push_back(gemm_time);
    time_vector.push_back(dequantize_time);
    // time_vector.push_back(int4gemm_time);
    // time_vector.push_back(fp16gemm_time);
    // time_vector.push_back(dequantize_time);
    time_vector.push_back(LSQ_time);
    time_vector.push_back(method1_time);
    time_vector.push_back(method2_time);
    time_vector.push_back(method3_time);
    // auto sample_x = torch::cat({sample_x1, sample_x2}, 0);

    return std::make_tuple(grad_input, grad_alpha, grad_output, time_vector, first_transform, second_transform, x1_len, x2_len, scale1);
    // return std::make_tuple(gemm1, gemm2, gemm3, gemm4, sum_y1_column, sum_y2_column);
}
